#include "hip/hip_runtime.h"
#ifndef _MCTS_GPU_CU_
#define _MCTS_GPU_CU_

#include "headers_gpu.h"

/*__constant__ short BitsSetTable256[256] =
 
 {
 0, 1, 1, 2, 1, 2, 2, 3, 1, 2, 2, 3, 2, 3, 3, 4,
 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
 3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
 1, 2, 2, 3, 2, 3, 3, 4, 2, 3, 3, 4, 3, 4, 4, 5,
 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
 3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
 2, 3, 3, 4, 3, 4, 4, 5, 3, 4, 4, 5, 4, 5, 5, 6,
 3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
 3, 4, 4, 5, 4, 5, 5, 6, 4, 5, 5, 6, 5, 6, 6, 7,
 4, 5, 5, 6, 5, 6, 6, 7, 5, 6, 6, 7, 6, 7, 7, 8
 };*/

#define CHECK_DIR(x,y,who) \
if (!IS_OCCUPIED(board, x, y))\
{\
dirs = moveGPU(board, x, y, who);\
if (IS_SET_DIR_ANY(dirs) && MAX_TASKS_STATIC > t.howMany)\
t.taskList[t.howMany++] = flipGPU(board, dirs, x, y, who);\
}

#define MAX_GPU_THREADS 1024
#define GROUP 32

void checkCUDAError(const char *msg);
__device__ short scoreGPU (node board, int player);
__global__ void MCTS(node* i_value, short *s, short player, short* random);
__host__ __device__ tasks getChildrenGPU (int who, node board);
__device__ __host__ node flipGPU (node board, int dirs, int x, int y, int player);
__host__ __device__ int moveGPU (node board, int x, int y, int player);
__device__ short rG(node startBoard, int startPlayer, short* randoms, short maxDepth, int randOffset, int master_player);
__device__ short rG2(node startBoard, int startPlayer, short* randoms, short maxDepth, int randOffset, int master_player);

__global__ void MCTS(node* i_value, short *s, short player, short* randoms) {
	
	const unsigned long total = gridDim.x * blockDim.x;
	const unsigned long tid = (blockDim.x * blockIdx.x + threadIdx.x);
    
	short startPlayer;//[MAX_GPU_THREADS];
	
	startPlayer = randoms[total+3 ];
	
	int depth = randoms[total+1];
	short master_player = randoms[total+2];
	
	//s[tid] = 0;
	
	s[tid] = rG2(i_value[blockIdx.x * blockDim.x], startPlayer, randoms, 60, 0, master_player);
    
	__syncthreads();
	
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}

__device__ short scoreGPU (node board, int player) {
	
    short score = 0;
    int x,y;
    /*	for (x = 0; x < 2; x++) {
     score +=  ((BitsSetTable256[board.blackOrWhite[player][x]] -   BitsSetTable256[board.blackOrWhite[!player][x]]) +
     (BitsSetTable256[board.blackOrWhite[player][x+2]] - BitsSetTable256[board.blackOrWhite[!player][x+2]])+
     (BitsSetTable256[board.blackOrWhite[player][x+4]] - BitsSetTable256[board.blackOrWhite[!player][x+4]])+
     (BitsSetTable256[board.blackOrWhite[player][x+6]] - BitsSetTable256[board.blackOrWhite[!player][x+6]]));
     }
     */
	for (x = 0; x < 8; x++)
        for (y = 0; y < 8; y++)
            score += BOARD_GET_SCORE(board, x, y, player) -  BOARD_GET_SCORE(board, x, y, !player);
    
	return score;
	
}
//#define SMEM
__device__ short rG2(node startBoard, int startPlayer, short* randoms, short maxDepth, int randOffset, int master_player) {
	
	
    const unsigned long tid = (blockDim.x * blockIdx.x + threadIdx.x);
    const unsigned long total = gridDim.x * blockDim.x;
    __shared__ short localrands[128];
    
#ifdef SMEM
	//__syncthreads();
    localrands[threadIdx.x] = randoms[tid];
    // __syncthreads();
#endif
    tasks temp;
    int rand;
    int result = 0;
    
    for (int i = 0; i < maxDepth; i++) {
        
        temp = getChildrenGPU(startPlayer, startBoard);
        
        if (temp.howMany > 0)
        {
            //put to shared mem
            //
#ifdef SMEM
            rand = localrands[(threadIdx.x + i) % blockDim.x] % temp.howMany;
#else
            rand = randoms[(tid + i) % total] % temp.howMany;
#endif
            startBoard = temp.taskList[rand];
            startPlayer = !startPlayer;
            __syncthreads();
        }
        
        
        else break;
        __syncthreads();
        
    }
    
	//if (scoreGPU(startBoard,master_player) > 0) result = 1;
    //result = scoreGPU(startBoard,master_player);
	return 64 - scoreGPU(startBoard,!master_player);
}

__device__ short rG(node startBoard, int startPlayer, short* randoms, short maxDepth, int randOffset, int master_player) {
	
	
    const unsigned long tid = (blockDim.x * blockIdx.x + threadIdx.x);
    const unsigned long total = gridDim.x * blockDim.x;
    __shared__ short localrands[128];
    
#ifdef SMEM
	//__syncthreads();
    localrands[threadIdx.x] = randoms[tid];
    // __syncthreads();
#endif
    tasks temp;
    int rand;
    int result = 0;
    
    for (int i = 0; i < maxDepth; i++) {
        
        temp = getChildrenGPU(startPlayer, startBoard);
        
        if (temp.howMany > 0)
        {
            //put to shared mem
            //
#ifdef SMEM
            rand = localrands[(threadIdx.x + i) % blockDim.x] % temp.howMany;
#else
            rand = randoms[(tid + i) % total] % temp.howMany;
#endif
            startBoard = temp.taskList[rand];
            startPlayer = !startPlayer;
            __syncthreads();
        }
        
        
        else break;
        __syncthreads();
        
    }
    
	if (scoreGPU(startBoard,master_player) > 0) result = 1;
    //result = scoreGPU(startBoard,master_player);
	return result;
}


__host__ tasks getChildrenGPU (int who, node board) {
	
	tasks t;
	t.howMany = 0;
	int dirs;
	int x;
    
	
	for (x = 0; x < 8; x++) {
        
        
		CHECK_DIR(x,0,who);
		CHECK_DIR(x,1,who);
        
		CHECK_DIR(x,2,who);
		CHECK_DIR(x,3,who);
        
		CHECK_DIR(x,4,who);
		CHECK_DIR(x,5,who);
        
		CHECK_DIR(x,6,who);
		CHECK_DIR(x,7,who);
        
	}
    
	return t;
    
}


__device__ __host__ node flipGPU (node board, int dirs, int x, int y, int player) {
	
	int i;
	
	BOARD_SET_PLAYER(board, x, y, player);
	
	if ( IS_SET_DIR_DOWN(dirs) )
        for (i = x; !BOARD_IS_SET(board, i + 1, y, player); i++)
            BOARD_FLIP(board, i + 1, y);
	
	if ( IS_SET_DIR_UP(dirs) )
        for (i = x; !BOARD_IS_SET(board, i - 1, y, player); i--)
            BOARD_FLIP(board, i - 1, y);
	
	if ( IS_SET_DIR_LEFT(dirs) )
        for (i = y; !BOARD_IS_SET(board, x, i - 1, player); i--)
            BOARD_FLIP(board, x, i - 1);
	
	if ( IS_SET_DIR_RIGHT(dirs) )
        for (i = y; !BOARD_IS_SET(board, x, i + 1, player); i++)
            BOARD_FLIP(board, x, i + 1);
	
	if ( IS_SET_DIR_DOWN_RIGHT(dirs) )
        for (i = 1; !BOARD_IS_SET(board, x + i, y + i, player); i++)
            BOARD_FLIP(board, x + i, y + i);
	
	if ( IS_SET_DIR_DOWN_LEFT(dirs) )
        for (i = 1; !BOARD_IS_SET(board, x + i, y - i, player); i++)
            BOARD_FLIP(board, x + i, y - i);
	
	if ( IS_SET_DIR_UP_RIGHT(dirs) )
        for (i = 1; !BOARD_IS_SET(board, x - i, y + i, player); i++)
            BOARD_FLIP(board, x - i, y + i);
	
	if ( IS_SET_DIR_UP_LEFT(dirs) )
        for (i = 1; !BOARD_IS_SET(board, x - i, y - i, player); i++)
            BOARD_FLIP(board, x - i, y - i);
    
	return board;
    
}

__host__ __device__ int moveGPU (node board, int x, int y, int player) {
	
	int dirs = 0;
	int i;
	
	//Left
	if (y > 1)
        if (BOARD_IS_SET(board, x, y-1, !player))
        {
            for (i = y; i > 1; i--)
            {
                if (BOARD_IS_SET(board, x,  i - 2, player)
                    && BOARD_IS_SET(board, x,  i - 1, !player))
                {
                    
                    SET_DIR_LEFT(dirs);
                    break;
                }
                else if (!BOARD_IS_SET(board, x,  i - 2, !player)
                         && BOARD_IS_SET(board, x,  i - 1, !player))
                    break;
            }
        }
    
	//Down left
	if (x < 6 && y > 1)
        if BOARD_IS_SET(board, x + 1,  y - 1, !player)
        {
            for (i = 0; ((x + i) < 6 && (y - i) > 2); i++)
            {
                if (BOARD_IS_SET(board, x + i + 2, y - i - 2, player)
                    && BOARD_IS_SET(board, x + i + 1, y - i - 1, !player))
                {
                    SET_DIR_DOWN_LEFT(dirs);
                    break;
                }
                else if (!(BOARD_IS_SET(board, x + i + 2, y - i - 2, !player)
                           && BOARD_IS_SET(board, x + i + 1, y - i - 1, !player)))
                    break;
            }
        }
	
	//down
	if (x < 6)
        if (BOARD_IS_SET(board, x+1, y, !player))
        {
            for (i = x; i < 6; i++)
            {
                if (BOARD_IS_SET(board, i+2, y, player) && BOARD_IS_SET(board, i+1, y, !player)) {
                    SET_DIR_DOWN(dirs);
                    break;
                }
                
                else if (!BOARD_IS_SET(board, i+2, y, !player) && BOARD_IS_SET(board, i+1, y, !player))
                    break;
            }
        }
    
    //Down right
	if (x < 6 && y < 6)
        
        if BOARD_IS_SET(board, x + 1,  y + 1, !player)
        {
            for (i = 0; ((x + i) < 6 && (y + i) < 6); i++)
            {
                if (BOARD_IS_SET(board, x + i + 2, y + i + 2, player)
                    && BOARD_IS_SET(board, x + i + 1, y + i + 1, !player))
                {
                    SET_DIR_DOWN_RIGHT(dirs);
                    break;
                }
                else if (!BOARD_IS_SET(board, x + i + 2, y + i + 2, !player)
                         && BOARD_IS_SET(board, x + i + 1, y + i + 1, !player))
                    break;
            }
        }
    
	//Right
	if (y < 6)
        if (BOARD_IS_SET(board, x,  y + 1, !player))
        {
            for (i = y; i < 6; i++)
            {
                if (BOARD_IS_SET(board, x,  i + 2, player)
                    && BOARD_IS_SET(board, x,  i + 1, !player))
                {
                    SET_DIR_RIGHT(dirs);
                    break;
                }
                else if (!BOARD_IS_SET(board, x,  i + 2, !player)
                         && BOARD_IS_SET(board, x,  i + 1, !player))
                    break;
            }
        }
	
    //Up right
	if (x > 1 && y < 6)
        if BOARD_IS_SET(board, x - 1,  y + 1, !player)
        {
            for (i = 0; ((x - i) > 1 && (y + i) < 6); i++)
            {
                if (BOARD_IS_SET(board, x - i - 2, y + i + 2, player)
                    && BOARD_IS_SET(board, x - i - 1, y + i + 1, !player))
                {
                    SET_DIR_UP_RIGHT(dirs);
                    break;
                }
                else if (!(BOARD_IS_SET(board, x - i - 2, y + i + 2, !player)
                           && BOARD_IS_SET(board, x - i - 1, y + i + 1, !player)))
                    break;
            }
        }
	
	//Up
	if (x > 1)
        if (BOARD_IS_SET(board, x-1, y, !player))
        {
            for (i = x; i > 1; i--)
            {
                if (BOARD_IS_SET(board, i-2, y, player) && BOARD_IS_SET(board, i-1, y, !player)) {
                    SET_DIR_UP(dirs);
                    break;
                }
                else if (!BOARD_IS_SET(board, i-2, y, !player)
                         && BOARD_IS_SET(board, i-1, y, !player))
                    break;
            }
        }
	
	//Up left
	if (x > 1 && y > 1)
        
        if BOARD_IS_SET(board, x - 1,  y - 1, !player)
        {
            for (i = 0; ((x - i) > 1 && (y - i) > 1); i++)
            {
                if (BOARD_IS_SET(board, x - i - 2, y - i - 2, player)
                    && BOARD_IS_SET(board, x - i - 1, y - i - 1, !player))
                {
                    SET_DIR_UP_LEFT(dirs);
                    break;
                }
                else if (!(BOARD_IS_SET(board, x - i - 2, y - i - 2, !player)
                           && BOARD_IS_SET(board, x - i - 1, y - i - 1, !player)))
                    break;
            }
        }
    /*
     */
	return dirs;
	
	
}

#endif
