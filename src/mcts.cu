#include "hip/hip_runtime.h"
#include "headers_gpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "mcts_gpu.cu"
/* helpful macro for catching CUDA errors */
#define CUDA_SAFE_CALL(call) do{\
hipError_t err = call ; \
if (hipSuccess != err){\
fprintf(stderr, "cuda error at %s:%d, %s\n",\
__FILE__, __LINE__, hipGetErrorString(err));\
}\
}while(0)


#define getAverage(node) 			((double)node->success)/((double)node->simulations)
#define calculateMCTSVal(node, C) 	(getAverage(node) + sqrt( (C * log(node->parent->simulations))/ log(node->simulations)))


void 			backPropagate(MCTSnode* leaf, long sim, long suc);
void 			initMCTSnode(MCTSnode **node, MCTSnode * parent, short masterPlayer, struct node_struct board, char* chars, short childNum);
MCTSnode* 		selectNode(MCTSnode* root, float C, MCTSnode* best);
long 			nodeCount(MCTSnode* root);

short randoms[MAX_RAND];

MCTSnode* MCTS_test (node board, short who, short masterPlayer, short depth, float maxTime, float C, struct timeval startTime, int id, int maxprocs) {
    
	node 							outBoard = board;
	struct timeval					currentTime;
	MCTSnode 						*root, *selectedNode, *tempNode;
	char 							chars[30] = "abcdefghijklmnopqrstuwvxyz";
	tasks							tempChildren;
	short							randTemp;
	short							simResult;
	short							timeLeft = 1;
	double							timeElapsed;
	int							i;
    
	root = (MCTSnode*) malloc (sizeof(MCTSnode));
	
	if (root == NULL) {
#ifndef CURSES
		printf("MCTS: Root malloc problem\n");
#endif
		exit(0);
	}
	
	initMCTSnode(&root, root, masterPlayer, board, NULL, 0);
    
	root->active = 1;
	
	selectedNode = root;
	
	
	while (timeLeft) {
        
        
		//////////////////////////////////
		//expand children
		//////////////////* helpful macro for catching CUDA errors */
#define CUDA_SAFE_CALL(call) do{\
hipError_t err = call ; \
if (hipSuccess != err){\
fprintf(stderr, "cuda error at %s:%d, %s\n",\
__FILE__, __LINE__, hipGetErrorString(err));\
}\
}while(0)
        /////////////////
		if (selectedNode->childrenCreated == 0) {
			
			tempChildren = getChildren (selectedNode->who, selectedNode->board);
			selectedNode->childrenNo = tempChildren.howMany;
			selectedNode->childrenCreated = 1;
			
			
			if (selectedNode->childrenNo > 0) {
                
				selectedNode->children = (MCTSnode*)calloc(selectedNode->childrenNo, sizeof (MCTSnode));
                
				for (i = 0; i < tempChildren.howMany; i++) {
                    
                    tempNode = ((MCTSnode*)(selectedNode->children+i));
                    
                    initMCTSnode(&tempNode, selectedNode, masterPlayer, tempChildren.taskList[i], chars, i);
                    
                    
				}
			}
            
		}
		
		//pick first random from selected
		if (selectedNode->childrenNo > 0) {
            
			randTemp = rand() % selectedNode->childrenNo;
			getChild(selectedNode, randTemp)->active = 1;
            
			//simulate
			if (id < maxprocs) {
				simResult = performRandomSimulation (getChild(selectedNode, randTemp)->board, getChild(selectedNode, randTemp)->who, depth, masterPlayer);
				
				
				getChild(selectedNode, randTemp)->simulations++;
				getChild(selectedNode, randTemp)->success += simResult;
				
				backPropagate(getChild(selectedNode, randTemp), 1, simResult);
			}
			
		}
		
		//checkTime
		gettimeofday(&currentTime, 0);
		timeElapsed = getTimeDiff(&startTime, &currentTime);
        
        
		//printf("id = %d, %f\n", id, timeElapsed);
		if (timeElapsed >= maxTime)
			timeLeft = 0;
		
		
		//selectNode for next iter
		
		selectedNode = selectNode(root, C, NULL);
		
		//time loop end
		
	}
	
	
	return root;
    
}

void setDevice(short device) {
    
    hipSetDevice(device);
    
}

MCTSnode* MCTS_GPU (node board, short who, short masterPlayer, short depth, float maxTime, float C, struct timeval startTime, int id, int maxprocs, short b, short t, short device) {
    
	node 							outBoard = board;
	struct timeval						currentTime;
	MCTSnode 						*root, *selectedNode, *tempNode;
	char 							chars[30] = "abcdefghijklmnopqrstuwvxyz";
	tasks							tempChildren;
	short							randTemp;
	short							simResult;
	short							timeLeft = 1;
	double							timeElapsed;
	short							gpuid = 0;
	short							blocks = b;
	short							thread_no = t;
	short							no_elements = blocks * thread_no;
	short							GPU_THREADS = no_elements;
	void							*hostIn, *hostRand, *hostOut, *iMem, *rMem, *oMem;
	short							i;
	int							totalSim = 0;
    
	
    //if (id == 0) { //dla pierwszego gpu
	if (blocks < 0) blocks = 1;
	if (thread_no < 0 || thread_no > 1024) thread_no = 128;
	root = (MCTSnode*) malloc (sizeof(MCTSnode));
	
	printf("dev = %d GPU B: %d, T: %d\n", device, blocks, thread_no);
	if (root == NULL) {
#ifndef CURSES
		printf("MCTS: Root malloc problem\n");
#endif
		exit(0);
	}
	
	initMCTSnode(&root, root, masterPlayer, board, NULL, 0);
    
	root->active = 1;
	
	selectedNode = root;
	
	//hipSetDevice (gpuid);
    
    CUDA_SAFE_CALL(hipHostMalloc( (void**) &(hostIn), no_elements * sizeof(node)));
    CUDA_SAFE_CALL(hipHostMalloc( (void**) &(hostRand), (no_elements+256) * sizeof(short)));
    CUDA_SAFE_CALL(hipHostMalloc( (void**) &(hostOut),  no_elements * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc( (void**) &(iMem),   no_elements  * sizeof(node)));
    CUDA_SAFE_CALL(hipMalloc( (void**) &(rMem),  (no_elements+256)  * sizeof(short)));
    CUDA_SAFE_CALL(hipMalloc( (void**) &(oMem), no_elements * sizeof(int)));
    
	dim3 threads(thread_no, 1, 1);
	dim3 grid(blocks,1,1);
    
	//for (int i = 0 ; i <  MAX_RAND; i++)
	//randoms[i][g] =
    
	for (int i = 0 ; i <  no_elements; i++)
		((short*)(hostRand))[i] = rand() % 32;//randoms[i];
    
	int step = 0;
	while (timeLeft)	 {
        
        //	printf("step: %d\n", ++step);
        
        //////////////////////////////////
		//expand children
		//////////////////////////////////
		if (selectedNode->childrenCreated == 0) {
			
			tempChildren = getChildren (selectedNode->who, selectedNode->board);
			selectedNode->childrenNo = tempChildren.howMany;
			selectedNode->childrenCreated = 1;
			
			
			if (selectedNode->childrenNo > 0) {
                
				selectedNode->children = (MCTSnode*)calloc(selectedNode->childrenNo, sizeof (MCTSnode));
                
				for (i = 0; i < tempChildren.howMany; i++) {
                    
                    tempNode = ((MCTSnode*)(selectedNode->children+i));
                    
                    initMCTSnode(&tempNode, selectedNode, masterPlayer, tempChildren.taskList[i], chars, i);
                    
                    
				}
			}
            
		}
		
		//pick first random from selected
		if (selectedNode->childrenNo > 0) {
            
			randTemp = rand() % selectedNode->childrenNo;
			getChild(selectedNode, randTemp)->active = 1;
            
			//simulate
			if (id < maxprocs) {
                
                if (step > 100)
                    simResult = performRandomSimulation (getChild(selectedNode, randTemp)->board, getChild(selectedNode, randTemp)->who, depth, masterPlayer); else {
                        
                        for (int b = 0; b < blocks; b++)
                            ((node*)(hostIn))[b*thread_no] = getChild(selectedNode, randTemp)->board;
                        ((short*)(hostRand))[no_elements + 3] = getChild(selectedNode, randTemp)->who;
                        ((short*)(hostRand))[no_elements + 1] =  depth;
                        ((short*)(hostRand))[no_elements + 2] = masterPlayer;
                        CUDA_SAFE_CALL( hipMemcpy( iMem , hostIn, sizeof(node)* no_elements, hipMemcpyHostToDevice));
                        
                        CUDA_SAFE_CALL( hipMemcpy( rMem , hostRand, sizeof(short) * (no_elements+256), hipMemcpyHostToDevice));
                        
                        
                        
                        //printf("kernel\n");
                        MCTS <<< grid, threads >>> ( ((node*)iMem),  (short*) (oMem) , (short)who, (short*)rMem);
                        checkCUDAError("kernel execution");
                        
                        CUDA_SAFE_CALL( hipMemcpy( hostOut, oMem, sizeof(short) * no_elements, hipMemcpyDeviceToHost));
                    }
                int res = 0;
                
                for (int i = 0; i < no_elements; i++) {
                    //result[tempIndex]  += ((int*)t_in->hostOut)[i];
                    res += ((short*)hostOut)[i];
                    
                }
                totalSim += no_elements;
                //		printf("%d\n", totalSim);
                //printf("%d - %d, %d\n", res, no_elements, totalSim);
				getChild(selectedNode, randTemp)->simulations+= no_elements;
				getChild(selectedNode, randTemp)->success += res;//simResult;
				
				backPropagate(getChild(selectedNode, randTemp), no_elements, res);
			}
			
		}
		
		//checkTime
		gettimeofday(&currentTime, 0);
		timeElapsed = getTimeDiff(&startTime, &currentTime);
        
        
		//printf("id = %d, %f %ld, %ld\n", id, timeElapsed, ++iter, nodeCount(root));
		if (timeElapsed >= maxTime)
			timeLeft = 0;
		
		
		//selectNode for next iter
		
		selectedNode = selectNode(root, C, NULL);
		
		//time loop end
		
	}
	//printf("total %d\n", totalSim);
	CUDA_SAFE_CALL(hipFree(iMem));
	CUDA_SAFE_CALL(hipFree(oMem));
	CUDA_SAFE_CALL(hipFree(rMem));
	CUDA_SAFE_CALL(hipHostFree(hostIn));
	CUDA_SAFE_CALL(hipHostFree(hostOut));
	CUDA_SAFE_CALL(hipHostFree(hostRand));
    //}
	return root;
    
}
#define MAX_BLOCKS 512
MCTSnode* MCTS_GPU_root (node board, short who, short masterPlayer, short depth, float maxTime, float C, struct timeval startTime, int id, int maxprocs, short b, short t, short device) {
    
    
	node 							outBoard = board;
	struct timeval						currentTime;
	MCTSnode 						*root[MAX_BLOCKS], *selectedNode[MAX_BLOCKS], *tempNode[MAX_BLOCKS];
	char 							chars[30] = "abcdefghijklmnopqrstuwvxyz";
	tasks							tempChildren[MAX_BLOCKS];
	short							randTemp[MAX_BLOCKS];
	short							simResult[MAX_BLOCKS];
	short							timeLeft = 1;
	double							timeElapsed;
	short							gpuid = 0;
	short							blocks = b;
	short							thread_no = t;
	short							no_elements = blocks * thread_no;
	short							GPU_THREADS = no_elements;
	void							*hostIn, *hostRand, *hostOut, *iMem, *rMem, *oMem;
	short							i;
	int							totalSim = 0;
    
	//hipSetDevice(device);
    //if (id == 0) { //dla pierwszego gpu
	if (blocks < 0) blocks = 1;
	if (thread_no < 0 || thread_no > 1024) thread_no = 128;
	
	printf("[R]GPU B: %d, T: %d\n", blocks, thread_no);
	
	for (int b = 0; b < blocks; b++) {
        root[b] = (MCTSnode*) malloc (sizeof(MCTSnode));
        
        
        if (root[b] == NULL) {
#ifndef CURSES
            printf("MCTS: Root malloc problem\n");
#endif
            exit(0);
        }
        
        initMCTSnode(&(root[b]), root[b], masterPlayer, board, NULL, 0);
        
        root[b]->active = 1;
        
        
        selectedNode[b] = root[b];
	}
    //	printf("a\n");
	//hipSetDevice (gpuid);
    
    CUDA_SAFE_CALL(hipHostMalloc( (void**) &(hostIn), no_elements * sizeof(node)));
    CUDA_SAFE_CALL(hipHostMalloc( (void**) &(hostRand), (no_elements+256) * sizeof(short)));
    CUDA_SAFE_CALL(hipHostMalloc( (void**) &(hostOut),  no_elements * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc( (void**) &(iMem),   no_elements  * sizeof(node)));
    CUDA_SAFE_CALL(hipMalloc( (void**) &(rMem),  (no_elements+256)  * sizeof(short)));
    CUDA_SAFE_CALL(hipMalloc( (void**) &(oMem), no_elements * sizeof(int)));
    
	dim3 threads(thread_no, 1, 1);
	dim3 grid(blocks,1,1);
    
	//for (int i = 0 ; i <  MAX_RAND; i++)
	//randoms[i][g] =
    
	for (int i = 0 ; i <  no_elements; i++)
		((short*)(hostRand))[i] = rand() % 32;//randoms[i];
    
	int step = 0;
	//printf("b\n");
	while (timeLeft)	 {
        
        //	printf("step: %d\n", ++step);
        
        //////////////////////////////////
		//expand children
		//////////////////////////////////
		for (int b = 0; b < blocks; b++)
            if (selectedNode[b]->childrenCreated == 0) {
                
                tempChildren[b] = getChildren (selectedNode[b]->who, selectedNode[b]->board);
                selectedNode[b]->childrenNo = tempChildren[b].howMany;
                selectedNode[b]->childrenCreated = 1;
                
                
                if (selectedNode[b]->childrenNo > 0) {
                    
                    selectedNode[b]->children = (MCTSnode*)calloc(selectedNode[b]->childrenNo, sizeof (MCTSnode));
                    
                    for (i = 0; i < tempChildren[b].howMany; i++) {
                        
                        tempNode[b] = ((MCTSnode*)(selectedNode[b]->children+i));
                        
                        initMCTSnode(&tempNode[b], selectedNode[b], masterPlayer, tempChildren[b].taskList[i], chars, i);
                        
                        
                    }
                }
                
            }
		
		//pick first random from selected
		
		if (selectedNode[0]->childrenNo)
		{
            
            for (int b = 0; b < blocks; b++) {
                if (selectedNode[b]->childrenNo) {
                    randTemp[b] = rand() % selectedNode[b]->childrenNo;
                    getChild(selectedNode[b], randTemp[b])->active = 1;
                }
            }
            //simulate
            if (id < maxprocs) {
                
                if (step > 100)
                    for (int b = 0; b < blocks; b++) {
                        if (selectedNode[b]->childrenNo)
                            simResult[b] = performRandomSimulation (getChild(selectedNode[b], randTemp[b])->board, getChild(selectedNode[b], randTemp[b])->who, depth, masterPlayer);
                    }
                
                else {
					
                    for (int b = 0; b < blocks; b++)
                        if (selectedNode[b]->childrenNo)
                            ((node*)(hostIn))[b*thread_no] = getChild(selectedNode[b], randTemp[b])->board;
                        else
                            ((node*)(hostIn))[b*thread_no] = getChild(selectedNode[0], randTemp[0])->board;
                    ((short*)(hostRand))[no_elements + 3] = getChild(selectedNode[0], randTemp[0])->who;
                    ((short*)(hostRand))[no_elements + 1] =  depth;
                    ((short*)(hostRand))[no_elements + 2] = masterPlayer;
                    CUDA_SAFE_CALL( hipMemcpy( iMem , hostIn, sizeof(node)* no_elements, hipMemcpyHostToDevice));
                    
                    CUDA_SAFE_CALL( hipMemcpy( rMem , hostRand, sizeof(short) * (no_elements+256), hipMemcpyHostToDevice));
                    
                    
                    
                    //printf("kernel\n");
                    MCTS <<< grid, threads >>> ( ((node*)iMem),  (short*) (oMem) , (short)who, (short*)rMem);
                    checkCUDAError("kernel execution");
                    
                    CUDA_SAFE_CALL( hipMemcpy( hostOut, oMem, sizeof(short) * no_elements, hipMemcpyDeviceToHost));
                }
                int res[MAX_BLOCKS];
                for (int b = 0; b < blocks; b++) {
                    res[b] = 0;
                    for (int i = 0; i < thread_no; i++) {
                        //result[tempIndex]  += ((int*)t_in->hostOut)[i];
                        res[b] += ((short*)hostOut)[b*thread_no+i];
                        
                    }
                }
                totalSim += no_elements;
                //		printf("%d\n", totalSim);
                //printf("%d - %d, %d\n", res, no_elements, totalSim);
                for (int b = 0; b < blocks; b++) {
                    
                    if (selectedNode[b]->childrenNo)  {
                        getChild(selectedNode[b], randTemp[b])->simulations+= thread_no;
                        getChild(selectedNode[b], randTemp[b])->success += res[b];//simResult[0];
                        backPropagate(getChild(selectedNode[b], randTemp[b]), thread_no, res[b]);
                        
                    }
                }
            }
            
		}
		
		//checkTime
		gettimeofday(&currentTime, 0);
		timeElapsed = getTimeDiff(&startTime, &currentTime);
        
        
		//printf("id = %d, %f %ld, %ld\n", id, timeElapsed, ++iter, nodeCount(root[0]));
		if (timeElapsed >= maxTime)
			timeLeft = 0;
		
		
		//selectNode for next iter
		for (int b = 0; b < blocks; b++)
			selectedNode[b] = selectNode(root[b], C, NULL);
		
		//time loop end
		
	}
	//printf("total %d\n", totalSim);
	CUDA_SAFE_CALL(hipFree(iMem));
	CUDA_SAFE_CALL(hipFree(oMem));
	CUDA_SAFE_CALL(hipFree(rMem));
	CUDA_SAFE_CALL(hipHostFree(hostIn));
	CUDA_SAFE_CALL(hipHostFree(hostOut));
	CUDA_SAFE_CALL(hipHostFree(hostRand));
    
	for (int i = 1 ; i <  blocks; i++) {
		
		for (int j = 0; j < root[0]->childrenNo; j++) {
            //printf("%d %d %d %d\n", i, j, (getChild(root[i],j)->success), (getChild(root[i],j)->simulations));
            (getChild(root[0],j)->success) += (double)(getChild(root[i],j)->success);
            (getChild(root[0],j)->simulations) += (double)(getChild(root[i],j)->simulations);
        }
        root[0]->simulations += root[i]->simulations;
        root[0]->success += root[i]->success;
        
		deleteMCTS(root[i]);
		delete(root[i]);
	}
	
	return root[0];
    
}

MCTSnode* MCTS (node board, short who, short masterPlayer, short depth, float maxTime, float C, struct timeval startTime, int id, int maxprocs) {
    
	node 							outBoard = board;
	struct timeval					currentTime;
	MCTSnode 						*root, *selectedNode, *tempNode;
	char 							chars[30] = "abcdefghijklmnopqrstuwvxyz";
	tasks							tempChildren;
	short							randTemp;
	short							simResult;
	short							timeLeft = 1;
	double							timeElapsed;
	short							i;
    
	root = (MCTSnode*) malloc (sizeof(MCTSnode));
	
	if (root == NULL) {
#ifndef CURSES
		printf("MCTS: Root malloc problem\n");
#endif
		exit(0);
	}
	
	initMCTSnode(&root, root, masterPlayer, board, NULL, 0);
    
	root->active = 1;
	
	selectedNode = root;
	
	while (timeLeft) {
        
        
		//////////////////////////////////
		//expand children
		//////////////////////////////////
		if (selectedNode->childrenCreated == 0) {
			
			tempChildren = getChildren (selectedNode->who, selectedNode->board);
			selectedNode->childrenNo = tempChildren.howMany;
			selectedNode->childrenCreated = 1;
			
			
			if (selectedNode->childrenNo > 0) {
                
				selectedNode->children = (MCTSnode*)calloc(selectedNode->childrenNo, sizeof (MCTSnode));
                
				for (i = 0; i < tempChildren.howMany; i++) {
                    
                    tempNode = ((MCTSnode*)(selectedNode->children+i));
                    
                    initMCTSnode(&tempNode, selectedNode, masterPlayer, tempChildren.taskList[i], chars, i);
                    
                    
				}
			}
            
		}
		
		//pick first random from selected
		if (selectedNode->childrenNo > 0) {
            
			randTemp = rand() % selectedNode->childrenNo;
			getChild(selectedNode, randTemp)->active = 1;
            
			//simulate
			if (id < maxprocs) {
				simResult = performRandomSimulation (getChild(selectedNode, randTemp)->board, getChild(selectedNode, randTemp)->who, depth, masterPlayer);
				
				
				getChild(selectedNode, randTemp)->simulations++;
				getChild(selectedNode, randTemp)->success += simResult;
				
				backPropagate(getChild(selectedNode, randTemp), 1, simResult);
			}
			
		}
		
		//checkTime
		gettimeofday(&currentTime, 0);
		timeElapsed = getTimeDiff(&startTime, &currentTime);
        
        
		//printf("id = %d, %f %ld, %ld\n", id, timeElapsed, ++iter, nodeCount(root));
		if (timeElapsed >= maxTime)
			timeLeft = 0;
		
		
		//selectNode for next iter
		
		selectedNode = selectNode(root, C, NULL);
		
		//time loop end
		
	}
	
	
	return root;
    
}
MCTSnode* selectNode(MCTSnode* root, float C, MCTSnode* best) {
    
	MCTSnode* tempNode = NULL;
	short i;
    
	if (best == NULL) best = root;
	
	if (root->active)
		for (i = 0; i < root->childrenNo; i++) {
            
			if (calculateMCTSVal(getChild(root,i),C) > calculateMCTSVal(best,C))
				best = getChild(root,i);
			
			tempNode = selectNode(getChild(root,i), C, best);
			
			if (tempNode != NULL) {
                
				if (calculateMCTSVal(tempNode,C) > calculateMCTSVal(best,C))
                    best = tempNode;
			}
            
		}
	
	return best;
    
}

long nodeCount(MCTSnode* root) {
    
	long no = 1;
	short i;
    
	if (root->active)
		for (i = 0; i < root->childrenNo; i++) {
            
			no += nodeCount(getChild(root,i));
		}
	
	return no;
    
}


void backPropagate(MCTSnode* leaf, long sim, long suc) {
    
    
	if (leaf->parent != leaf) {
		leaf->parent->simulations += sim;
		leaf->parent->success += suc;
		backPropagate(leaf->parent, sim, suc);
	}
    
}

void deleteMCTS (MCTSnode* root) {
    
	short i;
	if (root)
        if (root->children != NULL) {
            
            for (i = 0; i < root->childrenNo; i++)
                deleteMCTS (getChild(root, i));
            
            free(root->children);
        }
    
}

void initMCTSnode(MCTSnode **node, MCTSnode * parent, short masterPlayer, struct node_struct board, char* chars, short childNum) {
    
    
	((MCTSnode*)*node)->simulations = 2;
	((MCTSnode*)*node)->success = 1;
	((MCTSnode*)*node)->children = NULL;
	((MCTSnode*)*node)->childrenCreated = 0;
	((MCTSnode*)*node)->childrenNo = 0;
	((MCTSnode*)*node)->parent = parent;
	((MCTSnode*)*node)->active = 0;
	((MCTSnode*)*node)->board = board;
	
	if (((MCTSnode*)*node) != parent) {
        
		((MCTSnode*)*node)->who = !(parent->who);
		((MCTSnode*)*node)->depth = parent->depth + 1;
		strncpy(((MCTSnode*)*node)->id, parent->id, ((MCTSnode*)*node)->depth);
		((MCTSnode*)*node)->id[((MCTSnode*)*node)->depth] = chars[childNum];
        
	}
	else {
        
		((MCTSnode*)*node)->who = masterPlayer;
		((MCTSnode*)*node)->depth = 0;
		((MCTSnode*)*node)->id[0] = 'R';
		((MCTSnode*)*node)->id[1] = '\0';	
        
	}
    
	
}

void printMCTSnode (MCTSnode* node, float C, short includeChildren) {
    
    short i;
#ifndef CURSES
	//printf("ID: %s, Who: %d, Depth: %d, Active: %d, %ld/%ld (%.2f), ChildrenNo: %d, Avg: %.2f, Val: %.2f\n", node->id, node->who, node->depth, node->active, node->success, node->simulations, (float)node->success/(float)node->simulations, node->childrenNo, getAverage(node), calculateMCTSVal(node, C));
	if (includeChildren) {
        
        //	printf("Children of %s:\n", node->id);
		
		for (i = 0; i < node->childrenNo; i++)
			printMCTSnode(getChild(node,i), C, 0);
        
	}
#endif
}

void MCTSTest(cmdArguments args, int argc, const char **argv) {
    
	struct timeval 		startTime, currentTime;
    node				board;
    int 				player, masterPlayer;
    
	gettimeofday(&startTime, 0);
    
#ifndef CURSES	
	printf("[MCTS Test Start] Time: %s\n", getCurrentLocalTime(&startTime));
#endif
	board = initBoard();
	player = START_PLAYER;
	masterPlayer = START_PLAYER;
	
	initRand ();
	MCTSnode* root = MCTS (board, player, masterPlayer, args.depth, args.maxTime, args.C, startTime, 0, 1); 
	
	printMCTSnode(root, args.C, 1);
	deleteMCTS(root);
	delete(root);
	gettimeofday(&currentTime, 0);
    
#ifndef CURSES
	printf("[MCTS Test End] Time: %s\n", getCurrentLocalTime(&currentTime));
#endif
}
